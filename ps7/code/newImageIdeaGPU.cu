#include "hip/hip_runtime.h"
#include <math.h>
#include <string.h>
#include <stdlib.h>
#include <stdio.h>
#include "ppmCU.h"

// Image from:
// http://7-themes.com/6971875-funny-flowers-pictures.html

// TODO: You must implement this
// The handout code is much simpler than the MPI/OpenMP versions
//__global__ void performNewIdeaIterationGPU( ... ) { ... }

// TODO: You should implement this
//__global__ void performNewIdeaFinalizationGPU( ... ) { ... }

// Perhaps some extra kernels will be practical as well?
//__global__ void ...GPU( ... ) { ... }

typedef struct {
     float red,green,blue;
} AccuratePixel;

typedef struct {
     int x, y;
     AccuratePixel *data;
} AccurateImage;

__global__ void convertImageToNewFormatGPU(PPMPixel* image, AccuratePixel* imageAccurate) {
    // Calculate data-index
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int w = gridDim.x * blockDim.x;
    int i = y * w + x;

    imageAccurate->data[i].red = (float) image->data[i].red;
    imageAccurate->data[i].green = (float) image->data[i].green;
    imageAccurate->data[i].blue = (float) image->data[i].blue;
}

// Convert a PPM image to a high-precision format 
AccurateImage *convertImageToNewFormat(PPMImage *image) {
    // Allocate space for PPM-image on device
    PPMPixel* devImageData;
    hipMalloc((void**) &devImageData, sizeof(PPMPixel) * image->x * image->y);
    // Copy image to device
    hipMemcpy(devImageData, image->data, image->x * image->y * sizeof(PPMPixel), hipMemcpyHostToDevice);
	// Make a copy
	AccurateImage *imageAccurate = createEmptyImage(image);
    // Allocate space for copy on device
    AccuratePixel* devAccurateImageData;
    hipMalloc((void**) devAccurateImageData, sizeof(AccuratePixel) * image->x * image->y);

    // Invoke kernel
    dim3 blocks(16, 16);
    dim3 threadsPerBlock(image->x/blocks.x, image->y/blocks.y);

    convertImageToNewFormatGPU<<<blocks, threadsPerBlock>>>(devImageData, devAccurateImageData);
    
    // Retrieve image from device
    hipMemcpy(imageAccurate->data, devAccurateImageData, image->x * image->y * sizeof(AccuratePixel), hipMemcpyDeviceToHost);
    // Cleanup
    hipFree(devImageData);
    hipFree(devAccurateImageData);

	imageAccurate->x = image->x;
	imageAccurate->y = image->y;
	
	return imageAccurate;
}

// Convert a high-precision format to a PPM image
PPMImage *convertNewFormatToPPM(AccurateImage *image) {
	// Make a copy
	PPMImage *imagePPM;
	imagePPM = (PPMImage *)malloc(sizeof(PPMImage));
	imagePPM->data = (PPMPixel*)malloc(image->x * image->y * sizeof(PPMPixel));
	for(int i = 0; i < image->x * image->y; i++) {
		imagePPM->data[i].red   = (unsigned char) image->data[i].red;
		imagePPM->data[i].green = (unsigned char) image->data[i].green;
		imagePPM->data[i].blue  = (unsigned char) image->data[i].blue;
	}
	imagePPM->x = image->x;
	imagePPM->y = image->y;
	
	return imagePPM;
}

AccurateImage *createEmptyImage(PPMImage *image){
	AccurateImage *imageAccurate;
	imageAccurate = (AccurateImage *)malloc(sizeof(AccurateImage));
	imageAccurate->data = (AccuratePixel*)malloc(image->x * image->y * sizeof(AccuratePixel));
	imageAccurate->x = image->x;
	imageAccurate->y = image->y;
	
	return imageAccurate;
}

// free memory of an AccurateImage
void freeImage(AccurateImage *image){
	free(image->data);
	free(image);
}

__global__ void performNewIdeaIterationGPU(AccuratePixel* imageInData, AccuratePixel* imageOutData, int size) {
    int centerX = blockIdx.x * blockDim.x + threadIdx.x;
    int centerY = blockIdx.y * blockDim.y + threadIdx.y;
    int w = gridDim.x * blockDim.x;
    int h = gridDim.y * blockDim.y;

    float sumR = 0;
    float sumG = 0;
    float sumB = 0;
    int countIncluded = 0;
    for (int x = -size; x <= size; x++) {
        int currentX = centerX + x;
        if (currentX < 0 || currentX >= w) {
            continue;
        };
        for (int y = -size; y <= size; y++) {
            int currentY = centerY + y;
            if(currentY < 0 || currentY >= h) {
                continue;
            }

            // Calculate index
            int i = currentY * w + currentX;
            // Accumulate
            sumR += imageInData[i].red;
            sumG += imageInData[i].green;
            sumB += imageInData[i].blue;
            countIncluded++;
        }
    }

    // Average color values
    float valueR = sumR / countIncluded;
    float valueG = sumG / countIncluded;
    float valueB = sumB / countIncluded;

    // Update outputimage
    int i = centerY * w + centerX;
    imageOutData[i].red = valueR;
	imageOutData[i].green = valueG;
	imageOutData[i].blue = valueB;
}

void performNewIdeaIteration(AccurateImage *imageOut, AccurateImage *imageIn, int size) {
	// Allocate space for imagedata on device
    AccuratePixel* devImageInData, devImageOutData;
    int* devSize;
    hipMalloc((void**) &devImageInData, sizeof(AccuratePixel) * imageIn->x * imageIn->y);
    hipMalloc((void**) &devImageOutData, sizeof(AccuratePixel) * imageIn->x * imageIn->y);
    hipMalloc((void**) &devSize, sizeof(int));
    // Copy image data to device
    hipMemcpy(devImageInData, imageIn->data, sizeof(AccuratePixel) * imageIn->x * imageIn->y, hipMemcpyHostToDevice);
    hipMemcpy(devSize, &size, sizeof(int), hipMemcpyHostToDevice);
	
    // Invoke kernel
    dim3 blocks(16, 16);
    dim3 threadsPerBlock(imageIn->x/blocks.x, imageIn->y/blocks.y);
    performNewIdeaIterationGPU<<<blocks, threadsPerBlock>>>(devImageInData, devImageOutData, devSize);

    // Retrieve image
    hipMemcpy(imageOut->data, devImageOutData, sizeof(AccuratePixel) * imageIn->x * imageIn->y, hipMemcpyDeviceToHost);
    hipFree(devImageInData);
    hipFree(devImageOutData);
    hipFree(devSize);
}

__device__ float threshold(float value) {
    if(value > 255.0f)
		return 255;
	else if (value < -1.0f) {
		value = 257.0f+value;
	    if(value > 255.0f)
			return 255;
		else
			return floorf(value);
	} else if (value > -1.0f && value < 0.0f) {
	    return 0;
	} else {
		return floorf(value);
	}
}

__global__ void performNewIdeaFinalizationGPU(
        AccuratePixel* imageInSmallData, AccuratPixel* imageInLargeData, PPMPixel* imageOutData) {
    // Calculate data-index
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int w = gridDim.x * blockDim.x;
    int i = y * w + x;

    float value = (imageInLarge->data[i].red - imageInSmall->data[i].red);
	imageOutData[i].red = threshold(value);

    float value = (imageInLarge->data[i].green - imageInSmall->data[i].green);
	imageOutData[i].green = threshold(value);

    float value = (imageInLarge->data[i].blue - imageInSmall->data[i].blue);
	imageOutData[i].blue = threshold(value);
}

// Perform the final step, and save it as a ppm in imageOut
void performNewIdeaFinalization(AccurateImage *imageInSmall, AccurateImage *imageInLarge, PPMImage *imageOut) {

	
	imageOut->x = imageInSmall->x;
	imageOut->y = imageInSmall->y;
	
    // Allocate device copies of image data
    AccuratePixel* devImageInSmallData, devImageInLargeData;
    PPMPixel* devImageOutData;
    hipMalloc((void**) devImageInSmallData, sizeof(AccuratePixel) * imageOut->x * imageOut->y);
    hipMalloc((void**) devImageInLargeData, sizeof(AccuratePixel) * imageOut->x * imageOut->y);
    hipMalloc((void**) devImageOutData, sizeof(PPMPixel) * imageOut->x * imageOut->y);
    // Copy image data to device
    hipMemcpy(devImageInSmallData, imageInSmall->data, sizeof(AccuratePixel) * imageOut->x * imageOut->y, hipMemcpyHostToDevice);
    hipMemcpy(devImageInLargeData, imageInLarge->data, sizeof(AccuratePixel) * imageOut->x * imageOut->y, hipMemcpyHostToDevice);

    // Invoke kernel
    dim3 blocks(16, 16);
    dim3 threadsPerBlock(imageOut->x/blocks.x, imageOut->y/blocks.y);
    performNewIdeaFinalizationGPU<<<blocks, threadsPerBlock>>>(devImageInSmallData, devImageInLargeData, devImageOutData);

    // Retrieve result
    hipMemcpy(imageOut->data, devImageOutData, sizeof(PPMPixel) * imagOut->x * imageOut->y, hipMemcpyDeviceToHost);
    hipFree(devImageInSmallData);
    hipFree(devImageInLargeData);
    hipFree(devImageOutData);
}

int main(int argc, char** argv) {
	
	PPMImage *image;
        
	if(argc > 1) {
		image = readPPM("flower.ppm");
	} else {
		image = readStreamPPM(stdin);
	}

    //int* GPUDevice;
    //hipGetDevice(GPUDevice);
    //struct hipDeviceProp_t* props;
    //hipGetDeviceProperties(props, *GPUDevice);
    //int maxThreadsPerBlock = props->maxThreadsPerBlock;

	AccurateImage *imageUnchanged = convertImageToNewFormat(image); // save the unchanged image from input image
	AccurateImage *imageBuffer = createEmptyImage(image);
	AccurateImage *imageSmall = createEmptyImage(image);
	AccurateImage *imageBig = createEmptyImage(image);
	
	PPMImage *imageOut;
	imageOut = (PPMImage *)malloc(sizeof(PPMImage));
	imageOut->data = (PPMPixel*)malloc(image->x * image->y * sizeof(PPMPixel));

	// Process the tiny case:
	performNewIdeaIteration(imageSmall, imageUnchanged, 2);
	performNewIdeaIteration(imageBuffer, imageSmall, 2);
	performNewIdeaIteration(imageSmall, imageBuffer, 2);
	performNewIdeaIteration(imageBuffer, imageSmall, 2);
	performNewIdeaIteration(imageSmall, imageBuffer, 2);
	
	// Process the small case:
	performNewIdeaIteration(imageBig, imageUnchanged,3);
	performNewIdeaIteration(imageBuffer, imageBig,3);
	performNewIdeaIteration(imageBig, imageBuffer,3);
	performNewIdeaIteration(imageBuffer, imageBig,3);
	performNewIdeaIteration(imageBig, imageBuffer,3);
	
	// save tiny case result
	performNewIdeaFinalization(imageSmall,  imageBig, imageOut);
	if(argc > 1) {
		writePPM("flower_tiny.ppm", imageOut);
	} else {
		writeStreamPPM(stdout, imageOut);
	}

	
	// Process the medium case:
	performNewIdeaIteration(imageSmall, imageUnchanged, 5);
	performNewIdeaIteration(imageBuffer, imageSmall, 5);
	performNewIdeaIteration(imageSmall, imageBuffer, 5);
	performNewIdeaIteration(imageBuffer, imageSmall, 5);
	performNewIdeaIteration(imageSmall, imageBuffer, 5);
	
	// save small case
	performNewIdeaFinalization(imageBig,  imageSmall,imageOut);
	if(argc > 1) {
		writePPM("flower_small.ppm", imageOut);
	} else {
		writeStreamPPM(stdout, imageOut);
	}

	// process the large case
	performNewIdeaIteration(imageBig, imageUnchanged, 8);
	performNewIdeaIteration(imageBuffer, imageBig, 8);
	performNewIdeaIteration(imageBig, imageBuffer, 8);
	performNewIdeaIteration(imageBuffer, imageBig, 8);
	performNewIdeaIteration(imageBig, imageBuffer, 8);

	// save the medium case
	performNewIdeaFinalization(imageSmall,  imageBig, imageOut);
	if(argc > 1) {
		writePPM("flower_medium.ppm", imageOut);
	} else {
		writeStreamPPM(stdout, imageOut);
	}
	
	// free all memory structures
	freeImage(imageUnchanged);
	freeImage(imageBuffer);
	freeImage(imageSmall);
	freeImage(imageBig);
	free(imageOut->data);
	free(imageOut);
	free(image->data);
	free(image);
	
	return 0;
}

