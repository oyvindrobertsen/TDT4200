#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <time.h>
#include "lodepng.h"

__global__ void invert_character(char* image) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    image[x] = ~image[x];
}

int main( int argc, char ** argv){

    size_t pngsize;
    unsigned char *png;
    const char * filename = "lenna512x512_inv.png";
    /* Read in the image */
    lodepng_load_file(&png, &pngsize, filename);

    unsigned char *image;
    unsigned int width, height;
    /* Decode it into a RGB 8-bit per channel vector */
    unsigned int error = lodepng_decode24(&image, &width, &height, png, pngsize);

    /* Check if read and decode of .png went well */
    if(error != 0){
        std::cout << "error " << error << ": " << lodepng_error_text(error) << std::endl;
    }

    // Setup timing
    float there, back_again;
    hipEvent_t start_1, start_2, end_1, end_2;
    hipEventCreate(&start_1);
    hipEventCreate(&start_2);
    hipEventCreate(&end_1);
    hipEventCreate(&end_2);

    char* dev_img;
    hipMalloc((void**)&dev_img, sizeof(char)*width*height*3);

    hipEventRecord(start_1, 0);
    hipMemcpy(dev_img, image, width*height*3*sizeof(char), hipMemcpyHostToDevice);
    hipEventRecord(end_1, 0);
    hipEventSynchronize(end_1);
    hipEventElapsedTime(&there, start_1, end_1);

    invert_character<<<512*3, 512>>>(dev_img);

    hipEventRecord(start_2, 0);
    hipMemcpy(image, dev_img, sizeof(char)*width*height*3, hipMemcpyDeviceToHost);
    hipEventRecord(end_2, 0);
    hipEventSynchronize(end_2);
    hipEventElapsedTime(&back_again, start_2, end_2);

    hipEventDestroy(start_1);
    hipEventDestroy(start_2);
    hipEventDestroy(end_1);
    hipEventDestroy(end_2);

    hipFree(dev_img);
    /* Save the result to a new .png file */
    lodepng_encode24_file("lenna512x512_orig.png", image , width,height);
    free(image);

    float total_mem_time = there + back_again;
    printf("Total time spent transferring data: %f\n", total_mem_time);

    return 0;
}

